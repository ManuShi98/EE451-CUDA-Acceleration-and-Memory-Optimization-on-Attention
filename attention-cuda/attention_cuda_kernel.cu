#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <vector>

const int BLOCK_SIZE = 32;

/*  input : batch_size * head * seq_len * input_dim
    q : batch_size * input_dim * dim_k
    k : batch_size * input_dim * dim_k
    v : batch_size * input_dim * dim_v
    
*/

// token_num and features can be divided by 32
template <typename scalar_t>
__global__ void matmul(
    torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> mat1,
    torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> mat2,
    torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> output,
    int batch_size, 
    int head_num, 
    int n, 
    int m,
    int k, 
    int limit) {
    int row = threadIdx.y;
    int col = threadIdx.x;
    float local = 0;
    int my_x = blockIdx.y*blockDim.y + threadIdx.y;
	int my_y = blockIdx.x*blockDim.x + threadIdx.x;	
    int batch = my_x/(head_num*((n+BLOCK_SIZE-1)/BLOCK_SIZE));
    int head = (my_x%(head_num*((n+BLOCK_SIZE-1)/BLOCK_SIZE)))/((n+BLOCK_SIZE-1)/BLOCK_SIZE);
    if (my_x*n+my_y >= limit) {
        return;
    }

    __shared__ float A_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_shared[BLOCK_SIZE][BLOCK_SIZE];

    for(int i = 0; i < (m+BLOCK_SIZE-1)/BLOCK_SIZE; i++) {
        A_shared[row][col] = mat1[batch][head][my_x][i*blockDim.y+col];
        B_shared[row][col] = mat2[batch][head][i*blockDim.x+row][my_y];
        __syncthreads();
        for(int j = 0; j < BLOCK_SIZE; j++){
            local+=A_shared[row][j]*B_shared[j][col];
        }
        __syncthreads();
    }
    output[batch][head][my_x][my_y] = local;
}

torch::Tensor block_matmul_cuda(
    torch::Tensor mat1,
    torch::Tensor mat2,
    int batch_size,
    int head_num,
    int n,
    int m,
    int k) {
    
    auto output = torch::empty({batch_size, head_num, n, k}, torch::CUDA(torch::kFloat));
    dim3 dimBlock = (BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid = (batch_size*head_num*((n+dimBlock.x-1)/dimBlock.x), batch_size*head_num*((k+dimBlock.y-1)/dimBlock.y));
    AT_DISPATCH_FLOATING_TYPES(mat1.type(), "attention_forward_cuda", ([&]{
        matmul<scalar_t><<<dimGrid, dimBlock>>>(
            mat1.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            mat2.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            output.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
            batch_size, 
            head_num, 
            n, 
            m,
            k,
            batch_size * head_num * n * k
        );
    }));
    return output;
}
